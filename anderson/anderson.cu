#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <thrust/complex.h>
#include <thrust/copy.h>
#include <hipblas.h>
#include "hip/hip_runtime_api.h"

using namespace std;

typedef thrust::complex<double> th_complex;

const int xmax = 256;
const int tmax = 500;
double dt = 0.02;
//float dx = 2*sqrt(dt);
double scale = 2.0;
double dx = 1.0;
th_complex imag_one (0.0, 1.0);
th_complex a = -imag_one*th_complex(dt/(dx*dx),0.0);
th_complex b = 1.0+a;

__global__ void cutrid_RC_1b(th_complex *a,th_complex *b,th_complex *c,th_complex *d,th_complex *x) {
  int idx_global=blockIdx.x*blockDim.x+threadIdx.x;
  int idx=threadIdx.x;

  __shared__ th_complex asub[255];
  __shared__ th_complex bsub[255];
  __shared__ th_complex csub[255];
  __shared__ th_complex dsub[255];

  asub[idx]=a[idx_global];
  bsub[idx]=b[idx_global];
  csub[idx]=c[idx_global];
  dsub[idx]=d[idx_global];
  __syncthreads();
  //Reduction
  for(int stride=1;stride<255;stride*=2) {
    int margin_left=(idx-stride);
    int margin_right=(idx+stride);
    if(margin_left<0) margin_left=0;
    if(margin_right>=255) margin_right=254;
    th_complex tmp1 = asub[idx] / bsub[margin_left];
    th_complex tmp2 = csub[idx] / bsub[margin_right];
    th_complex tmp3 = dsub[margin_right];
    th_complex tmp4 = dsub[margin_left];
    __syncthreads();

    dsub[idx] = dsub[idx] - tmp4*tmp1-tmp3*tmp2;
    bsub[idx] = bsub[idx]-csub[margin_left]*tmp1-asub[margin_right]*tmp2;

    tmp3 = -csub[margin_right];
    tmp4 = -asub[margin_left];

    __syncthreads();
    asub[idx] = tmp3*tmp1;
    csub[idx] = tmp4*tmp2;
    __syncthreads();
  }

  x[idx_global]=dsub[idx]/bsub[idx];

}

__global__ void altCU(th_complex* d_u, th_complex* d_uH, th_complex* d_V,
            int xmax, th_complex a, th_complex b) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if ((i<xmax-1)&(i>0)) {
    th_complex mod_rs[201];  //modified right side
    th_complex c[201];  //super-diagonal vector
    //calculate h_c
    for(int j = 0 ; j < xmax ; j++) {
      c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
    }
    //modify h_c
    c[0] /= b - d_V[i*xmax + 0];	//delime strednou diagonalou
    for(int j = 1 ; j < xmax ; j++) {
      c[j] /= (b - d_V[i*xmax + j]) + a/2.0*c[j-1];	//spodna diagonala v matici je -a/2 preto +
    }

    mod_rs[0]  = (1.0-a)*d_uH[i*xmax + 0] + a/2.0*(d_uH[(i-1)*xmax + 0]+d_uH[(i+1)*xmax + 0]);
    mod_rs[0] /= b - d_V[i*xmax + 0];
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di  = (1.0-a)*d_uH[i*xmax + j] + a/2.0*(d_uH[(i-1)*xmax + j]+d_uH[(i+1)*xmax + j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b - d_V[i*xmax + j])+a/2.0*c[j-1]);
    }
    d_u[i*xmax + xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      d_u[i*xmax + j]=mod_rs[j]-c[j]*d_u[i*xmax + j+1];

    }
  }
}

__global__ void transposeCU(th_complex* d_u,
                            th_complex* d_uH,int xmax) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if ((i < xmax)&(j < xmax)) {
    d_uH[i*xmax+j] = d_u[j*xmax+i];
  }
}
void printInitialVariables(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax], th_complex h_c[]);
void printResult(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax], th_complex h_c[]);
void initializeHostArrays(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax], th_complex h_c[]);
void stdDev_r(ofstream& r, float t, th_complex u[][xmax]);
void transpose(th_complex arr[][xmax]);
void altCPUa(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b);
void altCPUb(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b);
void altCPUc(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b);


th_complex h_u[xmax][xmax] = {}; //alocating on heap
th_complex h_uH[xmax][xmax] = {};
th_complex h_V[xmax][xmax] = {};
th_complex h_c[xmax] = {};
th_complex h_b[xmax] = {};
th_complex h_a[xmax] = {};
th_complex h_d[xmax] = {};
th_complex h_x[xmax] = {};

int main() {

  if(xmax > 1024) {printf("Size of arr is greater than maximal number of threads %i\n", xmax);}


  float arrSize = sizeof(th_complex) * xmax * xmax;
  float vektSize = sizeof(th_complex) * xmax;

  initializeHostArrays(h_u, h_uH, h_V, h_c);

  //////////////////////////////////////////////
  ////  CUDA                                ////
  //////////////////////////////////////////////
  /*
  ofstream r_foutCU("and_stdDev_rCU.dat");

  th_complex *d_u, *d_uH, *d_V, *d_c;
  th_complex *d_b, *d_d, *d_a, *d_x;
  hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);
  const hipDoubleComplex* _alpha = &alpha;
  const hipDoubleComplex* _beta = &beta;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipMalloc(&d_u, arrSize);
  hipMalloc(&d_uH, arrSize);
  hipMalloc(&d_V, arrSize);
  hipMalloc(&d_c, vektSize);
  hipMalloc(&d_a, vektSize);
  hipMalloc(&d_b, vektSize);
  hipMalloc(&d_d, vektSize);
  hipMalloc(&d_x, vektSize);
  checkCudaErrors(hipMemcpy(d_u, h_u, arrSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_uH, h_uH, arrSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_V, h_V, arrSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_c, h_c, vektSize, hipMemcpyHostToDevice));
  dim3 dimBlock(32,32);
  dim3 dimGrid(xmax / 32 + 1, xmax / 32 + 1);

  hipDoubleComplex* d_uc = reinterpret_cast<hipDoubleComplex* >(d_u);
  hipDoubleComplex* d_uHc = reinterpret_cast<hipDoubleComplex* >(d_uH);
  hipDoubleComplex* d_Vc = reinterpret_cast<hipDoubleComplex* >(d_V);

  for(int i = 0; i < xmax; i++) {
    h_c[i] = -a/2.0;
    h_a[i] = -a/2.0;
  }
  hipMemcpy(d_a, h_a, vektSize, hipMemcpyHostToDevice);
  hipMemcpy(d_c, h_c, vektSize, hipMemcpyHostToDevice);
  for (int t = 0; t< tmax; t++) {
    hipMemcpy(h_uH, d_u, arrSize, hipMemcpyDeviceToHost);
    for(int i = 1 ; i < xmax ; i++) {
      for(int j = 0 ; j < xmax ; j++) {
        h_b[j] = b - h_V[i][j];
        h_d[j] = (1.0-a)*h_uH[i][0] + a/2.0*(h_uH[i-1][0]+h_uH[i+1][0]);
      }
      hipMemcpy(d_b, h_b, vektSize, hipMemcpyHostToDevice);
      hipMemcpy(d_d, h_d, vektSize, hipMemcpyHostToDevice);
      cutrid_RC_1b<<<xmax,xmax>>>(d_a, d_b, d_c, d_d, d_x);
      hipMemcpy(h_x, d_x, vektSize, hipMemcpyDeviceToHost);
      for(int j = 0 ; j < xmax ; j++) {
        h_u[i][j] = h_x[j];
      }
    }
    hipMemcpy(d_u, h_u, arrSize, hipMemcpyHostToDevice);
    transpose(h_V);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_Vc, xmax,
                _beta, d_Vc, xmax,
                d_uHc, xmax);
    hipMemcpy(d_V, d_uH, arrSize, hipMemcpyDeviceToDevice);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_uc, xmax,
                _beta, d_uc, xmax,
                d_uHc, xmax);


    // hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
    //             _alpha, d_Vc, xmax,
    //             _beta, d_Vc, xmax,
    //             d_uHc, xmax);
    // hipMemcpy(d_V, d_uH, arrSize, hipMemcpyDeviceToDevice);
    // hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
    //             _alpha, d_uc, xmax,
    //             _beta, d_uc, xmax,
    //             d_uHc, xmax);

    if (t%100==0) {
      hipMemcpy(h_u, d_u, arrSize, hipMemcpyDeviceToHost);
      stdDev_r(r_foutCU,t,h_u);
    }
  }

  hipMemcpy(h_u, d_u, arrSize, hipMemcpyDeviceToHost);
  printResult(h_u, h_uH, h_V, h_c);
  hipFree(d_u);
  hipFree(d_V);
  hipFree(d_uH);
  hipFree(d_c);
*/
  //////////////////////////////////////////////
  ////  CPU                                 ////
  //////////////////////////////////////////////
  ofstream r_foutCPU("and_stdDev_rCPU.dat");
  for (int t = 0; t< tmax; t++) {
    altCPUa(h_u, h_V, h_c, xmax, a, b);
    transpose(h_u);
    transpose(h_V);
    altCPUa(h_u, h_V, h_c, xmax, a, b);
    transpose(h_u);
    transpose(h_V);
    if (t%100==0) {
      stdDev_r(r_foutCPU,t,h_u);
    }
  }
  printResult(h_u, h_uH, h_V, h_c);

  return 0;
}

void initializeHostArrays(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax], th_complex h_c[]) {
  for(int i = 0; i<xmax; i++){
    for (int j = 0; j < xmax; j++) {
      h_u[i][j] =th_complex(0.0, 0.0);
      h_uH[i][j]=th_complex(0.0, 0.0);
      h_V[i][j] =2*scale*(double)(rand()%10000/10000.0-0.5);
      h_V[i][j] *= dt/imag_one;
      //h_V[i][j] = 0;
    }
  }
  //Nastavenie pociatocnych podmienok
  h_u[xmax/2][xmax/2] = th_complex(1.0, 0);
  h_uH[xmax/2][xmax/2] = th_complex(1.0, 0);
}
void printInitialVariables(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax], th_complex h_c[]) {
  cout << "     dx== " << dx <<endl<< "     dt== " << dt << endl << "      a== " << a << endl;
  cout << "      b== " << b << endl;
}
void printResult(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax],th_complex h_c[]) {
  printInitialVariables(h_u, h_uH, h_V, h_c);
  ofstream fout("data.dat");
  double sum = 0;
  for(int i = 0; i<xmax; i++){
    for (int j = 0; j < xmax; j++) {
      double probability = h_u[i][j].real()*h_u[i][j].real();
      probability += h_u[i][j].imag()*h_u[i][j].imag();
      fout << i << " " << j << " " <<  probability << " " << endl;
      sum += probability;
    }
  }
  fout.close();
  cout << h_u[xmax/2][xmax/2] << endl;
  cout << "sum == " << sum << endl;
}
void stdDev_r(ofstream& r, float t, th_complex u[][xmax]) {
  th_complex sum = 0;
  for(int i = 0; i< xmax; i++) {
    for(int j = 0; j < xmax; j++) {
      sum += (float)(pow((float)(i-xmax/2),2) + pow((float)(j-xmax/2),2))*(u[i][j].real()*u[i][j].real() + u[i][j].imag()*u[i][j].imag());
    }
  }
  r << t/tmax << " " << sum.real() << endl;
}
void transpose(th_complex arr[][xmax]) {
  th_complex help;
  for(int i = 0; i < xmax; i++) {
    for(int j = i+1; j < xmax; j++) {
      help = arr[i][j];
      arr[i][j] = arr[j][i];
      arr[j][i] = help;
    }
  }
}
//Potencial na lavej
void altCPUa(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b) {

  th_complex mod_rs[xmax];  //modified right side

  for(int i = 0 ; i < xmax ; i++) {
    for (int j = 0 ; j < xmax ; j++) {
      h_uH[i][j] = h_u[i][j]; //This is preserved state in time = t
    }
  }

  for(int i = 1; i<xmax-1; i++) {
    //calculate h_c
    for(int j = 0 ; j < xmax ; j++) {
      h_c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
    }
    //modify h_c
    h_c[0] /= b - h_V[i][0];	//delime strednou diagonalou
    for(int j = 1 ; j < xmax ; j++) {
      h_c[j] /= (b - h_V[i][j]) + a/2.0*h_c[j-1];	//spodna diagonala v matici je -a/2 preto +
    }

    mod_rs[0]  = (1.0-a)*h_uH[i][0] + a/2.0*(h_uH[i-1][0]+h_uH[i+1][0]);
    mod_rs[0] /= b - h_V[i][0];
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di  = (1.0-a)*h_uH[i][j] + a/2.0*(h_uH[i-1][j]+h_uH[i+1][j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b - h_V[i][j])+a/2.0*h_c[j-1]);
    }
    h_u[i][xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      h_u[i][j]=mod_rs[j]-h_c[j]*h_u[i][j+1];
    }
  }
  cout.precision(17);
  //Kontrola ci okrajove body v mriezke su = 0
  for(int i = 0 ; i < xmax ; i++) {
    if(h_u[i][0].real() != 0.0) {cout << "warning h_u[i][0] ==" << fixed << h_u[i][0].real() << endl;}
    if(h_u[i][0].imag() != 0.0) {cout << "warning h_u[i][0] ==" << fixed << h_u[i][0].imag() << endl;}

    if(h_u[i][xmax-1].real() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}
    if(h_u[i][xmax-1].imag() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}

    if(h_u[0][i].real() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}
    if(h_u[0][i].imag() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}

    if(h_u[xmax-1][i].real() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
    if(h_u[xmax-1][i].imag() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
  }
}
//potencial na pravej
void altCPUb(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b) {

  th_complex mod_rs[xmax];  //modified right side

  for(int i = 0 ; i < xmax ; i++) {
    for (int j = 0 ; j < xmax ; j++) {
      h_uH[i][j] = h_u[i][j]; //This is preserved state in time = t
    }
  }
  //calculate h_c
  for(int j = 0 ; j < xmax ; j++) {
    h_c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
  }
  //modify h_c
  h_c[0] /= b;	//delime strednou diagonalou
  for(int j = 1 ; j < xmax ; j++) {
    h_c[j] /= (b) + a/2.0*h_c[j-1];	//spodna diagonala v matici je -a/2 preto +
  }

  for(int i = 1; i<xmax-1; i++) {

    mod_rs[0]  = h_V[i][0] + (1.0-a)*h_uH[i][0] + a/2.0*(h_uH[i-1][0]+h_uH[i+1][0]);
    mod_rs[0] /= b;
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di = h_V[i][j] + (1.0-a)*h_uH[i][j] + a/2.0*(h_uH[i-1][j]+h_uH[i+1][j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b)+a/2.0*h_c[j-1]);
    }
    h_u[i][xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      h_u[i][j]=mod_rs[j]-h_c[j]*h_u[i][j+1];
    }
  }
  cout.precision(17);
  //Kontrola ci okrajove body v mriezke su = 0
  for(int i = 0 ; i < xmax ; i++) {
    if(h_u[i][0].real() != 0.0) {cout << "warning h_u[i][0] ==" << fixed << h_u[i][0].real() << endl;}
    if(h_u[i][0].imag() != 0.0) {cout << "warning h_u[i][0] ==" << fixed << h_u[i][0].imag() << endl;}

    if(h_u[i][xmax-1].real() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}
    if(h_u[i][xmax-1].imag() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}

    if(h_u[0][i].real() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}
    if(h_u[0][i].imag() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}

    if(h_u[xmax-1][i].real() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
    if(h_u[xmax-1][i].imag() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
  }
}
//potencial na oboch
void altCPUc(th_complex h_u[][xmax], th_complex h_V[][xmax], th_complex h_c[],
            int xmax, th_complex a, th_complex b) {

  th_complex mod_rs[xmax];  //modified right side

  for(int i = 0 ; i < xmax ; i++) {
    for (int j = 0 ; j < xmax ; j++) {
      h_uH[i][j] = h_u[i][j]; //This is preserved state in time = t
    }
  }

  for(int i = 1; i<xmax-1; i++) {
    //calculate h_c
    for(int j = 0 ; j < xmax ; j++) {
      h_c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
    }
    //modify h_c
    h_c[0] /= b - 0.5*h_V[i][0];	//delime strednou diagonalou
    for(int j = 1 ; j < xmax ; j++) {
      h_c[j] /= (b - 0.5*h_V[i][j]) + a/2.0*h_c[j-1];	//spodna diagonala v matici je -a/2 preto +
    }

    mod_rs[0]  = 0.5*h_V[i][0] + (1.0-a)*h_uH[i][0] + a/2.0*(h_uH[i-1][0]+h_uH[i+1][0]);
    mod_rs[0] /= b - 0.5*h_V[i][0];
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di = 0.5*h_V[i][j] + (1.0-a)*h_uH[i][j] + a/2.0*(h_uH[i-1][j]+h_uH[i+1][j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b - 0.5*h_V[i][j])+a/2.0*h_c[j-1]);
    }
    h_u[i][xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      h_u[i][j]=mod_rs[j]-h_c[j]*h_u[i][j+1];
    }
  }
  cout.precision(17);
  //Kontrola ci okrajove body v mriezke su = 0
  for(int i = 0 ; i < xmax ; i++) {
    if(h_u[i][0].real() != 0.0) {cout << setprecision(10) << "warning h_u[i][0] ==" << fixed << h_u[i][0].real() << endl;}
    if(h_u[i][0].imag() != 0.0) {cout << setprecision(10) << "warning h_u[i][0] ==" << fixed << h_u[i][0].imag() << endl;}

    if(h_u[i][xmax-1].real() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}
    if(h_u[i][xmax-1].imag() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}

    if(h_u[0][i].real() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}
    if(h_u[0][i].imag() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}

    if(h_u[xmax-1][i].real() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
    if(h_u[xmax-1][i].imag() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
  }
}
