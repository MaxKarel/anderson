#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cmath>
#include <fstream>
#include <stdio.h>
#include <thrust/complex.h>
#include <thrust/copy.h>
#include <hipblas.h>
#include "hipsolver.h"
#include "hip/hip_runtime_api.h"

using namespace std;

typedef thrust::complex<double> th_complex;

const int xmax = 201;
const int tmax = 150000;
double dt = 0.002;
//float dx = 2*sqrt(dt);
double scale = 2.0;
double dx = 1.0;
th_complex imag_one (0.0, 1.0);
th_complex a = -imag_one*th_complex(dt/(dx*dx),0.0);
th_complex b = 1.0+a;

__global__ void altCU(th_complex* d_u, th_complex* d_uH, th_complex* d_V,
            int xmax, th_complex a, th_complex b) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if ((i<xmax-1)&(i>0)) {
    th_complex mod_rs[201];  //modified right side
    th_complex c[201];  //super-diagonal vector
    //calculate h_c
    for(int j = 0 ; j < xmax ; j++) {
      c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
    }
    //modify h_c
    c[0] /= b - d_V[i*xmax + 0];	//delime strednou diagonalou
    for(int j = 1 ; j < xmax ; j++) {
      c[j] /= (b - d_V[i*xmax + j]) + a/2.0*c[j-1];	//spodna diagonala v matici je -a/2 preto +
    }

    mod_rs[0]  = (1.0-a)*d_uH[i*xmax + 0] + a/2.0*(d_uH[(i-1)*xmax + 0]+d_uH[(i+1)*xmax + 0]);
    mod_rs[0] /= b - d_V[i*xmax + 0];
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di  = (1.0-a)*d_uH[i*xmax + j] + a/2.0*(d_uH[(i-1)*xmax + j]+d_uH[(i+1)*xmax + j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b - d_V[i*xmax + j])+a/2.0*c[j-1]);
    }
    d_u[i*xmax + xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      d_u[i*xmax + j]=mod_rs[j]-c[j]*d_u[i*xmax + j+1];

    }
  }
}

__global__ void transposeCU(th_complex* d_u,
                            th_complex* d_uH,int xmax) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if ((i < xmax)&(j < xmax)) {
    d_uH[i*xmax+j] = d_u[j*xmax+i];
  }
}
void printInitialVariables(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]);
void printResult(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]);
void initializeHostArrays(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]);
void stdDev_r(ofstream& r, float t, th_complex u[][xmax]);
void transpose(th_complex arr[][xmax]);
void altCPU(th_complex h_u[][xmax], th_complex h_V[][xmax],
            int xmax, th_complex a, th_complex b);

  th_complex h_u[xmax][xmax] = {}; //alocating on heap
  th_complex h_uH[xmax][xmax] = {};
  th_complex h_V[xmax][xmax] = {};
  th_complex h_B[xmax][xmax] = {};
  th_complex h_d[xmax] = {};

int main() {

  if(xmax > 1024) {printf("Size of arr is greater than maximal number of threads %i\n", xmax);}


  float arrSize = sizeof(th_complex) * xmax * xmax;
  float vektSize = sizeof(th_complex) * xmax;

  initializeHostArrays(h_u, h_uH, h_V);

  //////////////////////////////////////////////
  ////  CUDA                                ////
  //////////////////////////////////////////////
  th_complex *d_u, *d_uH, *d_V, *d_B, *d_d;
  hipMalloc(&d_u, arrSize);
  hipMalloc(&d_uH, arrSize);
  hipMalloc(&d_V, arrSize);
  hipMalloc(&d_B, arrSize);
  hipMalloc(&d_d, vektSize);
  hipDoubleComplex* d_uc = reinterpret_cast<hipDoubleComplex* >(d_u);
  hipDoubleComplex* d_uHc = reinterpret_cast<hipDoubleComplex* >(d_uH);
  hipDoubleComplex* d_Vc = reinterpret_cast<hipDoubleComplex* >(d_V);
  hipDoubleComplex* d_Bc = reinterpret_cast<hipDoubleComplex* >(d_B);
  hipDoubleComplex* d_dc = reinterpret_cast<hipDoubleComplex* >(d_d);

  hipDoubleComplex alpha = make_hipDoubleComplex(1.0,0.0);
  hipDoubleComplex beta = make_hipDoubleComplex(0.0,0.0);
  const hipDoubleComplex* _alpha = &alpha;
  const hipDoubleComplex* _beta = &beta;

  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipsolverStatus_t status;
  hipsolverHandle_t handle_solver;
  hipsolverDnCreate(&handle_solver);


  ofstream r_fout("and_stdDev_rCU.dat");

  checkCudaErrors(hipMemcpy(d_u, h_u, arrSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_uH, h_uH, arrSize, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_V, h_V, arrSize, hipMemcpyHostToDevice));
  dim3 dimBlock(32,32);
  dim3 dimGrid(xmax / 32 + 1, xmax / 32 + 1);

  hipDoubleComplex* d_work;
  int work_size = 0;
  int info = 0;
  int* d_info;
  int* d_Ipiv;
  hipMalloc(&d_Ipiv,(xmax)*sizeof(int));
  hipMalloc(&d_info,sizeof(int));
  hipMalloc(&d_work,work_size*sizeof(hipDoubleComplex));
  hipsolverDnZgetrf_bufferSize(handle_solver,xmax,xmax,d_Bc,xmax,&work_size);
  cout.precision(25);
  for(int m = 0 ; m < xmax ; m++) {
    for(int n = 0 ; n < xmax ; n++) {
      h_B[m][n] = 0.0;
    }
  }
  for (int t = 0; t < tmax; t++) {
    /*for(int i = 0; i < xmax ; i++) {
      h_B[0][0] = b - h_V[i][0];
      h_B[0][1] = -a/2.0;
      for(int k = 1 ; k < xmax -1 ; k++) {
        h_B[k][k-1] = -a/2.0;
        h_B[k][k] = b - h_V[i][k];
        h_B[k][k+1] = -a/2.0;
      }
      h_B[xmax-1][xmax-2] = -a/2.0;
      h_B[xmax-1][xmax-1] = b - h_V[i][xmax-1];
      hipMemcpy(d_B, h_B, arrSize, hipMemcpyHostToDevice);
      for(int j = 0 ; j < xmax ; j++) {
        h_d[j] = (1.0-a)*h_uH[i][j] + a/2.0*(h_uH[i-1][j]+h_uH[i+1][j]);
      }
      hipMemcpy(d_d, h_d, vektSize, hipMemcpyHostToDevice);

      hipsolverDnZgetrf(handle_solver,xmax,xmax,d_Bc,xmax,d_work,d_Ipiv,d_info);
      hipMemcpy(&info, d_info,sizeof(int),hipMemcpyDeviceToHost);
    	if(info!=0){
    			cout << info << endl;
    		}
      status = hipsolverDnZgetrs(handle_solver,HIPBLAS_OP_N ,xmax,1,d_Bc,xmax,d_Ipiv,d_dc,xmax,d_info);
    	if(status!=HIPSOLVER_STATUS_SUCCESS){
    		cout << status << endl;
    	}
      hipMemcpy(d_u+i ,d_d,vektSize,hipMemcpyDeviceToDevice);
      hipDeviceSynchronize();
    }

    transpose(h_V);*/
    altCU<<<1,xmax>>>(d_u, d_uH, d_V, xmax, a, b);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_Vc, xmax,
                _beta, d_Vc, xmax,
                d_uHc, xmax);
    hipMemcpy(d_V, d_uH, arrSize, hipMemcpyDeviceToDevice);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_uc, xmax,
                _beta, d_uc, xmax,
                d_uHc, xmax);

    altCU<<<1,xmax>>>(d_u, d_uH, d_V, xmax, a, b);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_Vc, xmax,
                _beta, d_Vc, xmax,
                d_uHc, xmax);
    hipMemcpy(d_V, d_uH, arrSize, hipMemcpyDeviceToDevice);
    hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, xmax, xmax,
                _alpha, d_uc, xmax,
                _beta, d_uc, xmax,
                d_uHc, xmax);
    //hipMemcpy(d_V, d_uH, arrSize, hipMemcpyDeviceToDevice);
    if (t%100==0) {
      hipMemcpy(h_u, d_u, arrSize, hipMemcpyDeviceToHost);
      stdDev_r(r_fout,t,h_u);
    }
  }

  hipMemcpy(h_u, d_u, arrSize, hipMemcpyDeviceToHost);
  printResult(h_u, h_uH, h_V);
  hipFree(d_u);
  hipFree(d_V);
  hipFree(d_uH);

  //////////////////////////////////////////////
  ////  CPU                                 ////
  //////////////////////////////////////////////
  /*ofstream r_fout("std.cpp");
  for (int t = 0; t< tmax; t++) {
    altCPU(h_u, h_V, xmax, a, b);
    transpose(h_u);
    transpose(h_V);
    altCPU(h_u, h_V, xmax, a, b);
    transpose(h_u);
    transpose(h_V);
    if (t%10==0) {
      stdDev_r(r_fout,t,h_u);
    }
  }
  printResult(h_u, h_uH, h_V, h_c);
  */

  return 0;
}

void initializeHostArrays(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]) {
  for(int i = 0; i<xmax; i++){
    for (int j = 0; j < xmax; j++) {
      h_u[i][j] =th_complex(0.0, 0.0);
      h_uH[i][j]=th_complex(0.0, 0.0);
      h_V[i][j] =th_complex(2*scale*(float)(rand()%10000/10000.0-0.5), 0.0);
      h_V[i][j] *= th_complex(dt,0.0)/imag_one;
      //h_V[i][j] = 0;
    }
  }
  //Nastavenie pociatocnych podmienok
  h_u[xmax/2][xmax/2] = th_complex(1.0, 0);
  h_uH[xmax/2][xmax/2] = th_complex(1.0, 0);
}
void printInitialVariables(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]) {
  cout << "     dx== " << dx <<endl<< "     dt== " << dt << endl << "      a== " << a << endl;
  cout << "      b== " << b << endl;
}
void printResult(th_complex h_u[][xmax], th_complex h_uH[][xmax], th_complex h_V[][xmax]) {
  printInitialVariables(h_u, h_uH, h_V);
  ofstream fout("data.dat");
  float sum = 0;
  for(int i = 0; i<xmax; i++){
    for (int j = 0; j < xmax; j++) {
      float probability = h_u[i][j].real()*h_u[i][j].real();
      probability += h_u[i][j].imag()*h_u[i][j].imag();
      fout << i << " " << j << " " <<  probability << " " << endl;
      sum += probability;
    }
  }
  fout.close();
  cout << h_u[xmax/2][xmax/2] << endl;
  cout << "sum == " << sum << endl;
}
void stdDev_r(ofstream& r, float t, th_complex u[][xmax]) {
  th_complex sum = 0;
  for(int i = 0; i< xmax; i++) {
    for(int j = 0; j < xmax; j++) {
      sum += (float)(pow((float)(i-xmax/2),2) + pow((float)(j-xmax/2),2))*(u[i][j].real()*u[i][j].real() + u[i][j].imag()*u[i][j].imag());
    }
  }
  r << t/tmax << " " << sum.real() << endl;
}
void transpose(th_complex arr[][xmax]) {
  th_complex help;
  for(int i = 0; i < xmax; i++) {
    for(int j = i+1; j < xmax; j++) {
      help = arr[i][j];
      arr[i][j] = arr[j][i];
      arr[j][i] = help;
    }
  }
}

void altCPU(th_complex h_u[][xmax], th_complex h_V[][xmax],
            int xmax, th_complex a, th_complex b) {

  th_complex mod_rs[xmax];  //modified right side
  th_complex h_c[xmax];
  for(int i = 0 ; i < xmax ; i++) {
    for (int j = 0 ; j < xmax ; j++) {
      h_uH[i][j] = h_u[i][j]; //This is preserved state in time = t
    }
  }

  for(int i = 1; i<xmax-1; i++) {
    //calculate h_c
    for(int j = 0 ; j < xmax ; j++) {
      h_c[j] = -a/2.0;	//spodna diagonala v matici, je pri \psi(t-\Delta)
    }
    //modify h_c
    h_c[0] /= b - h_V[i][0];	//delime strednou diagonalou
    for(int j = 1 ; j < xmax ; j++) {
      h_c[j] /= (b - h_V[i][j]) + a/2.0*h_c[j-1];	//spodna diagonala v matici je -a/2 preto +
    }

    mod_rs[0]  = (1.0-a)*h_uH[i][0] + a/2.0*(h_uH[i-1][0]+h_uH[i+1][0]);
    mod_rs[0] /= b - h_V[i][0];
    th_complex di;  //unmodified right side, help variable
    for(int j=1; j < xmax-1; j++) {
      di  = (1.0-a)*h_uH[i][j] + a/2.0*(h_uH[i-1][j]+h_uH[i+1][j]);
      mod_rs[j] = (di+a/2.0*mod_rs[j-1])/((b - h_V[i][j])+a/2.0*h_c[j-1]);
    }
    h_u[i][xmax-1]=0; //mod_rs[j];
    for(int j=xmax-2; j>0; j--) {
      h_u[i][j]=mod_rs[j]-h_c[j]*h_u[i][j+1];
    }
  }
  cout.precision(17);
  //Kontrola ci okrajove body v mriezke su = 0
  for(int i = 0 ; i < xmax ; i++) {
    if(h_u[i][0].real() != 0.0) {cout << setprecision(10) << "warning h_u[i][0] ==" << fixed << h_u[i][0].real() << endl;}
    if(h_u[i][0].imag() != 0.0) {cout << setprecision(10) << "warning h_u[i][0] ==" << fixed << h_u[i][0].imag() << endl;}

    if(h_u[i][xmax-1].real() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}
    if(h_u[i][xmax-1].imag() != 0.0) {cout << "warning h_u[i][xmax-1] ==" << h_u[i][xmax-1] << endl;}

    if(h_u[0][i].real() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}
    if(h_u[0][i].imag() != 0.0) {cout << "warning h_u[0][i] ==" << h_u[0][i] << endl;}

    if(h_u[xmax-1][i].real() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
    if(h_u[xmax-1][i].imag() != 0.0) {cout << "warning h_u[xmax-1][i] ==" << h_u[xmax-1][i] << endl;}
  }
}
